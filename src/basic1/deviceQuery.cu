
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * query the gpu info
 */
int main1(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);//传递deviceCount 的内存地址
	std::cout<<"device count:"<<deviceCount<<std::endl;
	for(int i=0;i<deviceCount;i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,i);
		std::cout<<"device ："<<i<<":"<<prop.name<<std::endl;
		std::cout<<"global mem ："<<prop.totalGlobalMem/1024/1024<<"MB"<<std::endl;
		std::cout<<"sm number ："<<prop.multiProcessorCount<<std::endl;
		std::cout<<"shared mem per thread block ："<<prop.sharedMemPerBlock/1024<<"KB"<<std::endl;
		std::cout<<"max thread num per thread block ："<<prop.maxThreadsPerBlock<<std::endl;
		std::cout<<"register num per thred block ："<<prop.regsPerBlock<<std::endl;
		std::cout<<"max thread num per multi processor ："<<prop.maxThreadsPerMultiProcessor<<std::endl;
		std::cout<<"max wrap per multi processor ："<<prop.maxThreadsPerMultiProcessor/32<<std::endl;
	}




	return 0;
}
