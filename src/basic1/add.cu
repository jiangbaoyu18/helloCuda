
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

__global__ void Plus(float A[],float B[],float C[],int n){
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	C[i]=A[i]+B[i];
}

int main(){
	struct timeval start ,end;
	gettimeofday(&start,NULL);
	float *A,*B,*C,*Ad,*Bd,*Cd;
	int n=1024*1024;
	int size=n*sizeof(float);

	A=(float*)malloc(size);
	B=(float*)malloc(size);
	C=(float*)malloc(size);

	for(int i=0;i<n;i++){
		A[i]=90;
		B[i]=10;
	}

	hipMalloc(&Ad,size);//传入Ad指针的地址，初始化Ad指针，指向分配的显存地址 void** 为了类型通用
	hipMalloc((void **)&Bd,size);
	hipMalloc((void **)&Cd,size);

	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,size,hipMemcpyHostToDevice);
	hipMemcpy(Cd,C,size,hipMemcpyHostToDevice);

	dim3 dimBlock(512);
	dim3 dimGrid(n/512); // 此处总共使用了1024 * 1024 个线程处理数据，每个数组元素对应一个thread，
	Plus<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);

	hipMemcpy(C,Cd,size,hipMemcpyDeviceToHost);

	float error=0;
	for(int i=0;i<n;i++){
		error=fabs(100-C[i]);
	}
	cout<<"error is  "<<error<<endl;

	free(A);
	free(B);
	free(C);
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	gettimeofday(&end,NULL);
	int timeuse=1000000*(end.tv_sec-start.tv_sec)+end.tv_usec-start.tv_usec;
	cout<<"time use:"<<timeuse/1000<<"ms"<<endl;


	return 0;
}
